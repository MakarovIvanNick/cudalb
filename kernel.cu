#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32 //16

void getInfoCUDADevice(hipDeviceProp_t& prop, int id) {
	printf("CUDA device %i name  - %s\n", id, prop.name);
	printf("CUDA device %i Warp size in threads  - %i\n", id, prop.warpSize);
	printf("CUDA device %i Maximum number of threads per block  - %i\n", id, prop.maxThreadsPerBlock);
	printf("CUDA device %i multiprocessors count  - %i\n", id, prop.multiProcessorCount);
	printf("CUDA device %i Maximum size of each dimension of a block  - %i %i %i\n", id, prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("CUDA device %i Maximum size of each dimension of a grid  - %i %i %i\n", id, prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

__global__ void matrixMult(const int64_t *A, const int64_t *B, int64_t *result, int size) {
	//printf("blockIdx.y = %d,blockIdx.x = %d, threadIdx.y = %d, threadIdx.x = %d\n", blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x);
	int bx = blockIdx.x;  
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int ia = size * (gridDim.y * by + ty);
	int ib = gridDim.x * bx + tx;
	int ic = ia + ib;
	
	int64_t sum = 0;
	
	for (int k = 0; k < size; k++) {
		sum += A[ia + k] * B[k * size + ib];
	}
	result[ic] = sum;
}

void printResultMatr(const int* matr, int size) {
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) {
			printf(" %d ", matr[i * size + j]);
		}
		printf("\n");
	}
}
void compareMatrix(const int64_t* f, const int64_t* s, int size) {
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) {
			if (f[i * size + j] != s[i * size + j]) {
				printf("Matrixes not equal!\n");
				return;
			}
		}
	}
	printf("Matrixes is equal!\n");
}

int main()
{
	int count;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&count);
	//printf("Count CUDA devices - %i\n", count);
	hipGetDeviceProperties(&prop, count - 1);
	getInfoCUDADevice(prop, count - 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	
	int size = 2048;
	for (int iter = 0; iter < 10; iter++) {
		printf("ex num: %d\n", iter);
		
		size_t byte_size = size * size * sizeof(int64_t);
		int64_t* h_A = (int64_t*)malloc(byte_size);
		int64_t* h_B = (int64_t*)malloc(byte_size);
		int64_t* h_C = (int64_t*)malloc(byte_size);
		int64_t* CPU_C = (int64_t*)malloc(byte_size);

		for (int i = 0; i < size * size; ++i) {
			h_A[i] = rand() % 100;
			h_B[i] = rand() % 100;
			CPU_C[i] = 0;
		}
		//CPU (or host)
		
		printf("Scalar: \n");
		hipEventRecord(start, 0);
		if (iter < 2) {
			for (int i = 0; i < size; ++i) {
				for (int j = 0; j < size; ++j) {
					for (int k = 0; k < size; ++k) {
						//Rs[i][j] += Am[i][k] * Bm[k][j];
						CPU_C[i * size + j] += h_A[i * size + k] * h_B[k * size + j];
					}
				}
			}
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float result_time_cpu;
		hipEventElapsedTime(&result_time_cpu, start, stop);
		printf("Time: %f milliseconds\n", result_time_cpu);
		
		// printResultMatr(CPU_C, size);
		//GPU (or device)
		printf("GPU: \n");

		int64_t* d_A = NULL;
		hipMalloc((void**)&d_A, byte_size);
		hipMemcpy(d_A, h_A, byte_size, hipMemcpyHostToDevice);

		int64_t* d_B = NULL;
		hipMalloc((void**)&d_B, byte_size);
		hipMemcpy(d_B, h_B, byte_size, hipMemcpyHostToDevice);

		int64_t* d_C = NULL;
		hipMalloc((void**)&d_C, byte_size);

		hipEventRecord(start, 0);

		const dim3 block(32, 32);
		const dim3 grid(size / block.x, size / block.y);
		matrixMult <<< grid, block >>> (d_A, d_B, d_C, size);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float result_time_gpu;
		hipEventElapsedTime(&result_time_gpu, start, stop);
		printf("Time: %f milliseconds\n", result_time_gpu);

		hipMemcpy(h_C, d_C, byte_size, hipMemcpyDeviceToHost);
		// printResultMatr(h_C, size);
		//compare
		compareMatrix(h_C, CPU_C, size);

		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		free(h_A);
		free(h_B);     
		free(h_C); 
		free(CPU_C);
	}
	hipEventDestroy(start);  
	hipEventDestroy(stop);

	return 0;
}