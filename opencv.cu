#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include "hip/hip_runtime.h"
#include <iostream>

__global__ void computeIntensity(const uchar* img1, const uchar* img2, uchar* result, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int offset = y * cols + x;

        float intensity = ((img1[offset * 3] + img1[offset * 3 + 1] + img1[offset * 3 + 2]) +
                          (img2[offset * 3] + img2[offset * 3 + 1] + img2[offset * 3 + 2])) / 6.0f;

        result[offset] = static_cast<uchar>(255 * intensity / 510);
    }
}

int main() {
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	
    cv::Mat image1 = cv::imread("../src/anime1280x960.jpg");
    cv::Mat image2 = cv::imread("../src/gora1280x960.jpg");
    if (image1.empty() || image2.empty()) {
        printf("Images loading error\n");
        return -1;
    }
    std::cout<<"image 1 size: "<<image1.size()<<" image 2 size: " << image2.size()<<"\n";
    for (int iter = 1; iter <= 10; iter++) {
        cv::cuda::GpuMat gpuImage1, gpuImage2, gpuResult;
        gpuImage1.upload(image1);
        gpuImage2.upload(image2);

        gpuResult.create(image1.size(), CV_8UC1);

        const dim3 block(32, 32);
        const dim3 grid((image1.cols + block.x - 1) / block.x, (image1.rows + block.y - 1) / block.y);
        hipEventRecord(start, 0);
        computeIntensity<<<grid, block>>>(gpuImage1.data, gpuImage2.data, gpuResult.data, image1.rows, image1.cols);
        hipEventRecord(stop, 0);
        hipDeviceSynchronize();
        float result_time_cpu;
	    hipEventElapsedTime(&result_time_cpu, start, stop);
	    printf("ex: %d, time: %f milliseconds\n", iter, result_time_cpu); 
        cv::Mat result;
        gpuResult.download(result);

        cv::imwrite("../res/ex"+std::to_string(iter)+".jpg", result);
    }
    return 0;
}
